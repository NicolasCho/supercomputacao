#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <thrust/extrema.h>

using namespace std;

int main(){
    int n = 2518;
    thrust::host_vector<double> vcpu(n);

    auto start = std::chrono::system_clock::now();

    for(int i=0;i<n;i++)
       std::cin>>vcpu[i];

    auto end = std::chrono::system_clock::now();
    auto elapsed = end - start;
    cout << "Alocação:" << elapsed.count() << '\n';


    start = std::chrono::system_clock::now();
    thrust::device_vector<double> vgpu(vcpu);
    end = std::chrono::system_clock::now();
    elapsed = end - start;
    cout << "Cópia:" << elapsed.count() << '\n';

    double sum_total = thrust::reduce(vgpu.begin(), vgpu.end(), (double) 0, thrust::plus<double>());

    cout << "Preço médio: " << sum_total/n << endl;

    double sum_ly = thrust::reduce(vgpu.end()-365, vgpu.end(), (double) 0, thrust::plus<double>());

    cout << "Preço médio ano último ano: " << sum_ly/365 << endl;

    double max_total = thrust::reduce(vgpu.begin(), vgpu.end(), (double) 0, thrust::maximum<double>());
    double min_total = thrust::reduce(vgpu.begin(), vgpu.end(), (double) 1000000, thrust::minimum<double>());

    double max_ly = thrust::reduce(vgpu.end()-365, vgpu.end(), (double) 0, thrust::maximum<double>());
    double min_ly = thrust::reduce(vgpu.end()-365, vgpu.end(), (double) 1000000, thrust::minimum<double>());

    cout << "max total: " << max_total << endl;
    cout << "min total: " << min_total << endl;
    cout << "max ano: " << max_ly << endl;
    cout << "min ano: " << min_ly << endl;
    return 0;
}