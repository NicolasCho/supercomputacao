#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <cmath>
#include <random>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}

// CRIE UMA FUNCTOR PARA CALCULAR A SQUARE
struct square
{    
    __device__
    float operator()(const float& x) {   //x e y são os vetores
           return (x*x);  
    }
};



// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<float> &v) {
    float result;
    // ... add Thrust calls
    // AQUI VAO AS CHAMADAS THRUST 
    result = std::sqrt(thrust::transform_reduce(v.begin(), v.end(), square(), 0, thrust::plus<float>())); 
    return result;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Faça um  vector em thrust 
    thrust::host_vector<float> v_cpu(n);


    // inicilize o  vector
    ts = steady_clock::now();

    thrust::generate(v_cpu.begin(), v_cpu.end(), rand);

    thrust::device_vector<float> v_gpu(v_cpu);

    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Calcule a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(v_gpu);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}