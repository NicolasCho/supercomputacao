#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>

struct filme {
    int id;
    int inicio;
    int fim;
    int duracao;
    int categoria;
};

int main() {
    int N, M;
    std::cin >> N >> M;

    thrust::host_vector<int> categories(N);
    thrust::host_vector<int> start_times(N);
    thrust::host_vector<int> end_times(N);
    thrust::host_vector<int> L(M);

    for (int i = 0; i < M; i++) {
        std::cin >> L[i];
    }

    for (int i = 0; i < N; i++) {
        std::cin >> start_times[i] >> end_times[i] >> categories[i];
    }


     thrust::device_vector<int> dp((N + 1) * (M + 1), 0);

     for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= M; j++) {
            int max_count = 0;
            for (int k = 0; k < i; k++) {
                if (categories[k] == j && end_times[k] <= start_times[i] &&
                    dp[(k * (M + 1)) + j - 1] + 1 <= L[j - 1]) {
                    max_count = thrust::max(max_count, thrust::raw_reference_cast<int>(dp[(k * (M + 1)) + j - 1]) + 1);
                } else {
                    max_count = thrust::max(max_count, thrust::raw_reference_cast<int>(dp[(k * (M + 1)) + j]));
                }
            }
            dp[(i * (M + 1)) + j] = max_count;
        }
    }

    int max_count = *thrust::max_element(dp.begin() + N * (M + 1) + 1, dp.begin() + (N + 1) * (M + 1));

    std::cout << "Número máximo de filmes que podem ser assistidos: " << max_count << std::endl;

    return 0;
}