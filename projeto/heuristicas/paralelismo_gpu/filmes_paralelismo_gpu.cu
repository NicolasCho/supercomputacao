#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <vector>

struct knapsack_functor
{
    int* start_times;
    int* end_times;
    int* categories;
    int* L;
    int* results;
    int num_films;
    int num_categories;

    knapsack_functor(int* _start_times, int* _end_times, int* _categories, int* _L, int* _results, int _num_films, int _num_categories)
        : start_times(_start_times), end_times(_end_times), categories(_categories), L(_L), results(_results), num_films(_num_films), num_categories(_num_categories) {}

    __host__ __device__
    int operator()(const thrust::tuple<int, int>& t)
    {
        int i = thrust::get<0>(t);
        int c = thrust::get<1>(t);

        int max_count = 0;
        for (int k = 0; k < i; k++) {
            if (end_times[k] <= start_times[i] && results[k * num_categories + c - 1] <= L[c - 1]) {
                max_count = max(max_count, results[k * num_categories + c - 1] + 1);
            } else {
                max_count = max(max_count, results[k * num_categories + c - 1]);
            }
        }
        return max_count;
    }
};

int knapsack(int num_films, int num_categories, int* start_times, int* end_times, int* categories, int* L)
{
    thrust::device_vector<int> d_start_times(start_times, start_times + num_films);
    thrust::device_vector<int> d_end_times(end_times, end_times + num_films);
    thrust::device_vector<int> d_categories(categories, categories + num_films);
    thrust::device_vector<int> d_L(L, L + num_categories + 1);

    thrust::device_vector<int> d_results(num_films * num_categories, 0);
    thrust::device_vector<int> d_updated_results(num_films * num_categories, 0);

    thrust::counting_iterator<int> film_iter(0);
    thrust::counting_iterator<int> cat_iter(0);

    thrust::zip_iterator<thrust::tuple<thrust::counting_iterator<int>, thrust::counting_iterator<int>>> iter =
        thrust::make_zip_iterator(thrust::make_tuple(film_iter, cat_iter));

    thrust::transform(iter, iter + (num_films * num_categories),
                      d_updated_results.begin(), knapsack_functor(thrust::raw_pointer_cast(d_start_times.data()),
                                                                   thrust::raw_pointer_cast(d_end_times.data()),
                                                                   thrust::raw_pointer_cast(d_categories.data()),
                                                                   thrust::raw_pointer_cast(d_L.data()),
                                                                   thrust::raw_pointer_cast(d_results.data()),
                                                                   num_films, num_categories));

    thrust::copy(d_updated_results.begin(), d_updated_results.end(), d_results.begin());

    int max_value = thrust::reduce(thrust::device, d_results.begin(), d_results.end(), 0, thrust::maximum<int>());

    return max_value;
}

int main(int argc, char* argv[])
{
    int N = 0;
    int M = 0;

    std::cin >> N >> M;

    // Vector initialization
    std::vector<int> start_times_dev(N);
    std::vector<int> end_times_dev(N);
    std::vector<int> categories_dev(N);
    std::vector<int> L_dev(M + 1); // Increase the size by 1 to account for category 0

    // Preencher vetor de categorias
    int n_filmes_categoria;
    for (int i = 0; i < M; i++) {
        std::cin >> n_filmes_categoria;
        L_dev[i + 1] = n_filmes_categoria; // Use indexing instead of push_back()
    }

    // Preencher vetores de horarios (inicio e fim)
    int inicio, fim, categoria;
    for (int i = 0; i < N; i++) {
        std::cin >> inicio;
        std::cin >> fim;
        std::cin >> categoria;
        start_times_dev[i] = inicio;     
        end_times_dev[i] = fim;          
        categories_dev[i] = categoria;   
    }

    int max_value = knapsack(N, M, start_times_dev.data(), end_times_dev.data(), categories_dev.data(), L_dev.data());

    std::cout << "Max Value: " << max_value << std::endl;

    return 0;
}
