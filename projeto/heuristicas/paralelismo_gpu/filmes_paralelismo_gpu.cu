#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <fstream>
#include <iostream>
using namespace std;
int main(){
  // Ler os dados do arquivo de entrada
  //N = numero de filmes;  M = numero de categorias
  int N = 0;
  int M = 0;

  cin >> N >> M;

   // Vetores cpu
  thrust::host_vector<int> start_times_dev(N);
  thrust::host_vector<int> end_times_dev(N);
  thrust::host_vector<int> categories_dev(M);

  // Preencher vetor de categorias
  int n_filmes_categoria;
  for (int i = 0; i < M; i++){
      cin >> n_filmes_categoria;
      categories_dev.push_back(n_filmes_categoria);
  }

  // Preencher vetores de horarios (inicio e fim)
  int inicio, fim;
  for (int i = 0; i < N; i++){
      cin >> inicio;
      cin >> fim;
      start_times_dev.push_back(inicio);
      end_times_dev.push_back(fim);
  }

  // Carregar os dados do arquivo de entrada na memória da GPU
  thrust::device_vector<int> start_times(start_times_dev);
  thrust::device_vector<int> end_times(end_times_dev);
  thrust::device_vector<int> categories(categories_dev);

  // Criar a matriz de programação dinâmica
  thrust::device_vector<int> dp((N+1) * (M+1), 0);

  // Inicializar a primeira linha da matriz com zeros
  thrust::fill(dp.begin(), dp.begin() + M + 1, 0);

  // Preencher a matriz com as soluções para subproblemas menores
  for (int i = 1; i <= N; i++) {
    for (int j = 1; j <= M; j++) {
      // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
      int max_count = 0;
      for (int k = 0; k < i; k++) {
        if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= categories[j-1]) {
          max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
        } else {
          max_count = max(max_count, dp[(k*(M+1)) + j]);
        }
      }
      dp[(i*(M+1)) + j] = max_count;
    }
  }

  // Encontrar o número máximo de filmes que podem ser assistidos
  int max_count = 0;
  for (int j = 1; j <= M; j++) {
    max_count = max(max_count, dp[(N*(M+1)) + j]);
  }

  // Escrever o resultado no arquivo de saída
  ofstream outputFile;
  string result_file = "resultado.txt";  // Caminho relativo ao script que chama a heuristica
  outputFile.open(result_file, ios::app);
  string result = to_string(max_count);
  outputFile << result << endl;
  return 0;

}
