#include "hip/hip_runtime.h"
%%writefile paralelismo_gpu.cu
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <omp.h>
#include <fstream>
using namespace std;
struct filme {
    int id;
    int inicio;
    int fim;
    int duracao;
    int categoria;
};

int main(int argc, char *argv[]){
    clock_t start, end;
    start = clock();

    //Recebe o nome da variavel testada e o nome do arquivo
    string variavel = argv[1]; 
    string arquivo = argv[2];


    int N, M;
    std::cin >> N >> M;

    thrust::host_vector<int> categories(N);
    thrust::host_vector<int> start_times(N);
    thrust::host_vector<int> end_times(N);
    thrust::host_vector<int> L(M);

    for (int i = 0; i < M; i++) {
        std::cin >> L[i];
    }

    for (int i = 0; i < N; i++) {
        std::cin >> start_times[i] >> end_times[i] >> categories[i];
    }


     thrust::device_vector<int> dp((N + 1) * (M + 1), 0);

     for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= M; j++) {
            int max_count = 0;
            for (int k = 0; k < i; k++) {
                if (categories[k] == j && end_times[k] <= start_times[i] &&
                    dp[(k * (M + 1)) + j - 1] + 1 <= L[j - 1]) {
                    max_count = thrust::max(max_count, thrust::raw_reference_cast<int>(dp[(k * (M + 1)) + j - 1]) + 1);
                } else {
                    max_count = thrust::max(max_count, thrust::raw_reference_cast<int>(dp[(k * (M + 1)) + j]));
                }
            }
            dp[(i * (M + 1)) + j] = max_count;
        }
    }

    int max_count = *thrust::max_element(dp.begin() + N * (M + 1) + 1, dp.begin() + (N + 1) * (M + 1));

    //std::cout << "Número máximo de filmes que podem ser assistidos: " << max_count << std::endl;


    end = clock();
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC);


    // Escreve novo resultado no csv
    // Métricas: número de filmes alocados, quantidade de horas ocupadas, tempo de execução
    ofstream outputFile;
    string result_file = "results.csv";  // Caminho relativo ao script que chama a heuristica
    outputFile.open(result_file, ios::app);
    string result = variavel + "," + to_string(max_count) + "," + "0" + 
        "," + to_string(time_taken) + "," + arquivo;
    outputFile << result << endl;
    return 0;
}